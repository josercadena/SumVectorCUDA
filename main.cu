#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void MatAdd (float* A, float* B, float* C, int N){

	int index = threadIdx.x;
	if (index < 5){
		C[index] = A[index] + B[index];
	}
}

int main(void){

	/*Set array size*/
	int N = 5;
	int size = N * sizeof(float);

	/*Define and initialize arrays in HOST*/
	float* h_A = (float *)malloc(size);
	float* h_B = (float *)malloc(size);
	float* h_C = (float *)malloc(size);

	for (int i = 0; i < N; i++){
		h_A[i] = i;
		h_B[i] = i;
		h_C[i] = 0;
	}

	/*Define and allocate arrays in DEVICE*/
	float* d_A;
	float* d_B;
	float* d_C;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	/*Copy arrays from HOST to DEVICE*/
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

	/*Define level of parallelism*/
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(1,1,1);

	/*Launch kernel and synchronize*/
	MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
	hipDeviceSynchronize();

	/*Copy output array from DEVICE to HOST*/
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	/*Free device memory*/
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

//	/*Print to console*/
//	cout << "A[] = ";
//	for (int i = 0; i < N; i++){
//		if (i == 0) {cout << "[";}
//		cout << h_A[i];
//		if (i == N-1) {cout << "]" << endl << endl;} else {cout << ", ";}
//	}
//
//	cout << "B[] = ";
//	for (int i = 0; i < N; i++){
//		if (i == 0) {cout << "[";}
//		cout << h_B[i];
//		if (i == N-1) {cout << "]" << endl << endl;} else {cout << ", ";}
//	}
//
//	cout << "C[] = ";
//	for (int i = 0; i < N; i++){
//		if (i == 0) {cout << "[";}
//		cout << h_C[i];
//		if (i == N-1) {cout << "]" << endl;} else {cout << ", ";}
//
//	}

	return 0;
}
